#include "hip/hip_runtime.h"
#include "app_helper.hpp"
#include "cuda_helper.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <stdexcept>
#include <limits>

template<int TBlocksize, typename T>
__device__
T reduce(int tid, T *x, int n) {

  __shared__ T sdata[TBlocksize];

  int i = blockIdx.x * TBlocksize + tid;

  sdata[tid] = 0;

  // --------
  // Level 1: grid reduce, reading from global memory
  // --------

  // reduce per thread with increased ILP by 4x unrolling sum.
  // the thread of our block reduces its 4 grid-neighbored threads and advances by grid-striding loop
  while (i+3*gridDim.x*TBlocksize < n) {
    sdata[tid] += x[i] + x[i+gridDim.x*TBlocksize] + x[i+2*gridDim.x*TBlocksize] + x[i+3*gridDim.x*TBlocksize];
    i += 4*gridDim.x*TBlocksize;
  }
  // doing the remaining blocks
  while(i<n) {
    sdata[tid] += x[i];
    i += gridDim.x * TBlocksize;
  }

  __syncthreads();

  // --------
  // Level 2: block + warp reduce, reading from shared memory
  // --------

#pragma unroll
  for(int bs=TBlocksize, bsup=(TBlocksize+1)/2;
      bs>1;
      bs=bs/2, bsup=(bs+1)/2) {
    if(tid < bsup && tid+bsup<TBlocksize) {
      sdata[tid] += sdata[tid + bsup];
    }
    __syncthreads();
  }

  return sdata[0];
}

template<int TBlocksize, int TMaxWarpNum, typename T>
__global__
void kernel_reduce(T* x, T* y, int n)
{
  T block_result = reduce<TBlocksize>(threadIdx.x, x, n);

  unsigned warpid,smid;
  asm("mov.u32 %0, %%smid;":"=r"(smid));//get SM id
  asm("mov.u32 %0, %%warpid;":"=r"(warpid));//get warp id within SM

  // store block result to gmem
  if (threadIdx.x == 0)
    y[smid * TMaxWarpNum + warpid] += block_result;
}

template<int TBlocksize, typename T>
__global__
void kernel_reduce_2(T* x, T* y, int n)
{
  T block_result = reduce<TBlocksize>(threadIdx.x, x, n);

  // store block result to gmem
  if (threadIdx.x == 0)
    atomicAdd(y, block_result);
}

template<typename T, int TRuns, int TBlocksize, int TMaxWarpNum>
void reduce(size_t n, int dev) {

  CHECK_CUDA( hipSetDevice(dev) );
  CHECK_CUDA( hipFree(0) ); // force context init (applies clocks before getting props)

  hipDeviceProp_t prop;
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );
  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev);
  hipEvent_t cstart, cend;
  CHECK_CUDA(hipEventCreate(&cstart));
  CHECK_CUDA(hipEventCreate(&cend));
  hipStream_t cstream;
  CHECK_CUDA(hipStreamCreate(&cstream));


  T* h_x = new T[n];
  T* x;
  T* y;
  T* z;
  CHECK_CUDA( hipMalloc(&x, n*sizeof(T)) );
  CHECK_CUDA( hipMalloc(&y, TMaxWarpNum*numSMs*sizeof(T)) );
  CHECK_CUDA( hipMalloc(&z, sizeof(T)) );
  for (int i = 0; i < n; i++) {
    h_x[i] = static_cast<T>(1);
  }
  CHECK_CUDA( hipMemcpy( x, h_x, n*sizeof(T), hipMemcpyHostToDevice) );


  dim3 blocks = (n-1)/TBlocksize+1;
  dim3 blocks_2 = (TMaxWarpNum*numSMs-1)/TBlocksize+1;

  std::cout << " "
            << std::setw(3) << 0
            << ", " << dev
            << ", " << prop.name
            << ", " << prop.major << '.' << prop.minor
            << ", " << prop.memoryClockRate/1000
            << ", " << prop.clockRate/1000
            << ", " << n
            << ", " << numSMs
            << ", " << blocks.x
            << ", " << blocks.x/numSMs
            << ", " << blocks.x
            << ", " << TBlocksize
            << ", " << TRuns
    ;

  float milliseconds = 0;
  float min_ms = std::numeric_limits<float>::max();

  // -- REPETITIONS --
  for(int r=0; r<TRuns; ++r) {
    CHECK_CUDA( hipDeviceSynchronize() );
    CHECK_CUDA(hipEventRecord(cstart, cstream));
    CHECK_CUDA(hipMemset(y, 0, TMaxWarpNum*numSMs*sizeof(T)));
    CHECK_CUDA(hipMemset(z, 0, sizeof(T)));

    kernel_reduce<TBlocksize, TMaxWarpNum><<<blocks, TBlocksize, 0, cstream>>>(x, y, n);
    kernel_reduce_2<TBlocksize><<<blocks_2, TBlocksize, 0, cstream>>>(y, z, TMaxWarpNum*numSMs);

    CHECK_CUDA( hipEventRecord(cend, cstream) );
    CHECK_CUDA( hipEventSynchronize(cend) );
    CHECK_CUDA( hipGetLastError() );
    CHECK_CUDA( hipEventElapsedTime(&milliseconds, cstart, cend) );
    if(milliseconds<min_ms)
      min_ms = milliseconds;
  }

  T result_gpu;
  CHECK_CUDA( hipMemcpy( &result_gpu, z, sizeof(T), hipMemcpyDeviceToHost) );
  // check result
  if( result_gpu != static_cast<T>(n) ) {
    std::cerr << "\n\n" << result_gpu << " != " << n << "\n";
    throw std::runtime_error("RESULT MISMATCH");
  }
  std::cout << ", " << min_ms << " ms"
            << ", " << n*sizeof(T)/min_ms*1e-6 << " GB/s"
            << "\n";

  delete[] h_x;
  CHECK_CUDA(hipFree(x));
  CHECK_CUDA(hipFree(y));
  CHECK_CUDA(hipFree(z));
  CHECK_CUDA(hipEventDestroy(cstart));
  CHECK_CUDA(hipEventDestroy(cend));
  CHECK_CUDA(hipStreamDestroy(cstream));

}

int main(int argc, const char** argv)
{

  static constexpr int REPETITIONS = 3;
  static constexpr int MAX_WARPS_PER_SM = 64;
  using DATA_TYPE = int;

  const int dev=0;
  unsigned int n1 = 0;
  unsigned int n2 = 0;
  if(argc>=2)
    n1 = atoi(argv[1]);
  if(n1<2)
    n1 = 1<<28;
  if(argc==3) // range
    n2 = atoi(argv[2]);
  if(n2<n1)
    n2 = n1;

  print_header("reduction-mono",n1,n2);

  for(unsigned n=n1; n<=n2; n<<=1) {
    reduce<DATA_TYPE, REPETITIONS, 64, MAX_WARPS_PER_SM>(n, dev);
    reduce<DATA_TYPE, REPETITIONS, 128, MAX_WARPS_PER_SM>(n, dev);
    reduce<DATA_TYPE, REPETITIONS, 256, MAX_WARPS_PER_SM>(n, dev);
    reduce<DATA_TYPE, REPETITIONS, 512, MAX_WARPS_PER_SM>(n, dev);
    reduce<DATA_TYPE, REPETITIONS, 1024, MAX_WARPS_PER_SM>(n, dev);
  }

  CHECK_CUDA( hipDeviceReset() );
  return 0;
}
