#include "hip/hip_runtime.h"
#include "app_helper.hpp"
#include "cuda_helper.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <vector>

template<typename T, unsigned int TRuns, unsigned int TBlocksize, unsigned int TMaxWarpNum>
void reduce(size_t n, unsigned int dev) {

  static_assert(TMaxWarpNum>0, "TMaxWarpNum>0");

  CHECK_CUDA( hipSetDevice(dev) );
  CHECK_CUDA( hipFree(0) ); // force context init (applies clocks before getting props)

  hipDeviceProp_t prop;
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );
  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev);
  hipEvent_t cstart, cend;
  CHECK_CUDA(hipEventCreate(&cstart));
  CHECK_CUDA(hipEventCreate(&cend));
  hipStream_t cstream;
  CHECK_CUDA(hipStreamCreate(&cstream));

  /* allocate memory for input data on the host */
  T* d_in;
  T* d_out;
  std::vector<T> h_vec(n);
  std::fill(h_vec.begin(), h_vec.end(), 1);
  CHECK_CUDA(hipMalloc(&d_in, n*sizeof(T)));
  CHECK_CUDA(hipMalloc(&d_out, sizeof(T)));
  CHECK_CUDA(hipMemcpy(d_in, h_vec.data(), n*sizeof(T), hipMemcpyHostToDevice));
  // Determine temporary device storage requirements
  void     *d_temp_storage = NULL;
  size_t   temp_storage_bytes = 0;
  hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, n);
  // Allocate temporary storage
  CHECK_CUDA(hipMalloc(&d_temp_storage, temp_storage_bytes));

  T result_gpu = 0;

  dim3 blocks = 2; //( ((n+1)/2)-1)/TBlocksize+1; // ceil(ceil(n/2.0)/TBlocksize)
  dim3 blocks_2 = (TMaxWarpNum*numSMs-1)/TBlocksize+1;

  std::cout << " "
            << std::setw(3) << 0
            << ", " << dev
            << ", " << prop.name
            << ", " << prop.major << '.' << prop.minor
            << ", " << prop.memoryClockRate/1000
            << ", " << prop.clockRate/1000
            << ", " << n
            << ", " << numSMs
            << ", " << blocks.x
            << ", " << blocks.x/numSMs
            << ", " << blocks.x
            << ", " << TBlocksize
            << ", " << TRuns
    ;

  float milliseconds = 0;
  float min_ms = std::numeric_limits<float>::max();

  // -- REPETITIONS --
  for(unsigned int r=0; r<TRuns; ++r) {
    CHECK_CUDA( hipDeviceSynchronize() );
    CHECK_CUDA(hipEventRecord(cstart, cstream));

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, n);

    CHECK_CUDA( hipEventRecord(cend, cstream) );
    CHECK_CUDA( hipEventSynchronize(cend) );
    CHECK_CUDA( hipGetLastError() );
    CHECK_CUDA( hipEventElapsedTime(&milliseconds, cstart, cend) );
    if(milliseconds<min_ms)
      min_ms = milliseconds;
  }

  // check result
  CHECK_CUDA(hipMemcpy(&result_gpu,d_out,sizeof(T),hipMemcpyDeviceToHost));
  if( result_gpu != static_cast<T>(n) ) {
    std::cerr << "\n\n" << result_gpu << " != " << n << "\n";
    throw std::runtime_error("RESULT MISMATCH");
  }
  std::cout << ", " << min_ms << " ms"
            << ", " << n*sizeof(T)/min_ms*1e-6 << " GB/s"
            << "\n";

  CHECK_CUDA(hipEventDestroy(cstart));
  CHECK_CUDA(hipEventDestroy(cend));
  CHECK_CUDA(hipStreamDestroy(cstream));
  CHECK_CUDA(hipFree(d_in));
  CHECK_CUDA(hipFree(d_out));
  CHECK_CUDA(hipFree(d_temp_storage));

}


int main(int argc, const char** argv)
{

  static constexpr unsigned int REPETITIONS = 5;
  static constexpr unsigned int MAX_WARPS_PER_SM = 64; // hardware specific
  using DATA_TYPE = unsigned;

  const int dev=0;
  unsigned int n1 = 0;
  unsigned int n2 = 0;
  if(argc>=2)
    n1 = atoi(argv[1]);
  if(n1<2)
    n1 = 1<<28;
  if(argc==3) // range
    n2 = atoi(argv[2]);
  if(n2<n1)
    n2 = n1;

  print_header("reduction-cub",n1,n2);

  try{
    for(unsigned n=n1; n<=n2; n<<=1) {
      // cub seems to always run 256 threads per block
      reduce<DATA_TYPE, REPETITIONS, 256 /* just for output */, MAX_WARPS_PER_SM>(n, dev);
    }
  }catch(std::runtime_error e){
    std::cerr << e.what() << "\n";
    CHECK_CUDA( hipDeviceReset() );
    return 1;
  }

  CHECK_CUDA( hipDeviceReset() );
  return 0;
}
