#include "hip/hip_runtime.h"
#include "app_helper.hpp"
#include "cuda_helper.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <stdexcept>
#include <limits>

template<typename T>
__global__
void kernel_saxpy(T *x, T *y, unsigned int n, T a) {

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int s = blockDim.x * gridDim.x;
	while( i+s*2 < n )
	{
		y[i] = a * x[i] + y[i];
		i += s;
		y[i] = a * x[i] + y[i];
		i += s;
		y[i] = a * x[i] + y[i];
		i += s;
	}
	while(	i < n  	)
	{
		y[i] = a * x[i] + y[i];
		i += s;
	}
}


template<typename T, unsigned int TRuns, unsigned int TBlocksize>
void saxpy(size_t n, int dev) {

  CHECK_CUDA( hipSetDevice(dev) );
  CHECK_CUDA( hipFree(0) ); // force context init (applies clocks before getting props)

  hipDeviceProp_t prop;
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );
  hipEvent_t cstart, cend;
  CHECK_CUDA(hipEventCreate(&cstart));
  CHECK_CUDA(hipEventCreate(&cend));
  hipStream_t cstream;
  CHECK_CUDA(hipStreamCreate(&cstream));


  const T a = static_cast<T>(42);
  T* h_x = new T[n];
  T* h_y = new T[n];
  T* h_z = new T[n];
  T* x;
  T* y;
  CHECK_CUDA( hipMalloc(&x, n*sizeof(T)) );
  CHECK_CUDA( hipMalloc(&y, n*sizeof(T)) );
  for (unsigned int i = 0; i < n; i++) {
    h_x[i] = static_cast<T>(1);
    h_y[i] = static_cast<T>(2);
  }
  CHECK_CUDA( hipMemcpy( x, h_x, n*sizeof(T), hipMemcpyHostToDevice) );

  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev);

  unsigned int blocks_i = numSMs;
  unsigned int blocks_n = (n-1)/TBlocksize+1;
  unsigned int i=0;
  // -- GRIDSIZE LOOP --
  do{
    blocks_i <<= 1; // starting with 2*numSMs blocks per grid
    if(blocks_i>blocks_n)
      blocks_i = blocks_n;

    std::cout << " "
              << std::setw(3) << i++
              << ", " << dev
              << ", " << prop.name
              << ", " << prop.major << '.' << prop.minor
              << ", " << prop.memoryClockRate/1000
              << ", " << prop.clockRate/1000
              << ", " << n
              << ", " << numSMs
              << ", " << blocks_i
              << ", " << blocks_i/numSMs
              << ", " << blocks_n
              << ", " << TBlocksize
              << ", " << TRuns
      ;

    float milliseconds = 0;
    float min_ms = std::numeric_limits<float>::max();

    // -- REPETITIONS --
    for(unsigned int r=0; r<TRuns; ++r) {
      CHECK_CUDA( hipMemcpy( y, h_y, n*sizeof(T), hipMemcpyHostToDevice) );
      CHECK_CUDA( hipDeviceSynchronize() );
      CHECK_CUDA( hipEventRecord(cstart, cstream));

      kernel_saxpy<<<blocks_i, TBlocksize, 0, cstream>>>(x, y, n, a);

      CHECK_CUDA( hipEventRecord(cend, cstream) );
      CHECK_CUDA( hipEventSynchronize(cend) );
      CHECK_CUDA( hipGetLastError() );
      CHECK_CUDA( hipEventElapsedTime(&milliseconds, cstart, cend) );
      if(milliseconds<min_ms)
        min_ms = milliseconds;
    }

    CHECK_CUDA( hipMemcpy( h_z, y, n*sizeof(T), hipMemcpyDeviceToHost) );
    // check result
    for(unsigned int k=0; k<n; ++k) {
      if( h_z[k] != 1*a+2 ) {
        std::cerr << "\n\n y[" << k << "] = " << h_z[k] << "\n";
        throw std::runtime_error("RESULT MISMATCH");
      }
    }
    std::cout << ", " << min_ms << " ms"
              << ", " << 3*n*sizeof(T)/min_ms*1e-6 << " GB/s"
              << "\n";

  }while( blocks_i < blocks_n );

  delete[] h_x;
  delete[] h_y;
  delete[] h_z;
  CHECK_CUDA(hipFree(x));
  CHECK_CUDA(hipFree(y));
  CHECK_CUDA(hipEventDestroy(cstart));
  CHECK_CUDA(hipEventDestroy(cend));
  CHECK_CUDA(hipStreamDestroy(cstream));

}

int main(int argc, const char** argv)
{

  static constexpr unsigned int REPETITIONS = 5;
  using DATA_TYPE = unsigned;

  const int dev=0;
  unsigned int n1 = 0;
  unsigned int n2 = 0;
  if(argc>=2)
    n1 = atoi(argv[1]);
  if(n1<2)
    n1 = 1<<28;
  if(argc==3) // range
    n2 = atoi(argv[2]);
  if(n2<n1)
    n2 = n1;

  print_header("saxpy-grid",n1,n2);

  for(unsigned n=n1; n<=n2; n<<=1) {
    saxpy<DATA_TYPE, REPETITIONS, 64>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 128>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 256>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 512>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 1024>(n, dev);
  }

  CHECK_CUDA( hipDeviceReset() );
  return 0;
}
