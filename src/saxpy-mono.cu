#include "hip/hip_runtime.h"
#include "app_helper.hpp"
#include "cuda_helper.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <stdexcept>
#include <limits>

template<typename T>
__global__
void kernel_saxpy(T *x, T *y, int n, T a) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<n)
    y[i] = a * x[i] + y[i];
}


template<typename T, int TRuns, int TBlocksize>
void saxpy(size_t n, int dev) {

  CHECK_CUDA( hipSetDevice(dev) );
  CHECK_CUDA( hipFree(0) ); // force context init (applies clocks before getting props)

  hipDeviceProp_t prop;
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );
  hipEvent_t cstart, cend;
  CHECK_CUDA(hipEventCreate(&cstart));
  CHECK_CUDA(hipEventCreate(&cend));
  hipStream_t cstream;
  CHECK_CUDA(hipStreamCreate(&cstream));


  const T a = static_cast<T>(42);
  T* h_x = new T[n];
  T* h_y = new T[n];
  T* h_z = new T[n];
  T* x;
  T* y;
  CHECK_CUDA( hipMalloc(&x, n*sizeof(T)) );
  CHECK_CUDA( hipMalloc(&y, n*sizeof(T)) );
  for (int i = 0; i < n; i++) {
    h_x[i] = static_cast<T>(1);
    h_y[i] = static_cast<T>(2);
  }
  CHECK_CUDA( hipMemcpy( x, h_x, n*sizeof(T), hipMemcpyHostToDevice) );

  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev);

  int blocks_n = (n-1)/TBlocksize+1;

  std::cout << " "
            << std::setw(3) << 0
            << ", " << dev
            << ", " << prop.name
            << ", " << prop.major << '.' << prop.minor
            << ", " << prop.memoryClockRate/1000
            << ", " << prop.clockRate/1000
            << ", " << n
            << ", " << numSMs
            << ", " << blocks_n
            << ", " << blocks_n/numSMs
            << ", " << blocks_n
            << ", " << TBlocksize
            << ", " << TRuns
    ;

  float milliseconds = 0;
  float min_ms = std::numeric_limits<float>::max();

  // -- REPETITIONS --
  for(int r=0; r<TRuns; ++r) {
    CHECK_CUDA( hipMemcpy( y, h_y, n*sizeof(T), hipMemcpyHostToDevice) );
    CHECK_CUDA( hipDeviceSynchronize() );
    CHECK_CUDA( hipEventRecord(cstart, cstream));

    kernel_saxpy<<<blocks_n, TBlocksize, 0, cstream>>>(x, y, n, a);

    CHECK_CUDA( hipEventRecord(cend, cstream) );
    CHECK_CUDA( hipEventSynchronize(cend) );
    CHECK_CUDA( hipGetLastError() );
    CHECK_CUDA( hipEventElapsedTime(&milliseconds, cstart, cend) );
    if(milliseconds<min_ms)
      min_ms = milliseconds;
  }

  CHECK_CUDA( hipMemcpy( h_z, y, n*sizeof(T), hipMemcpyDeviceToHost) );
  // check result
  for(int k=0; k<n; ++k) {
    if( h_z[k] != 1*a+2 ) {
      std::cerr << "\n\n y[" << k << "] = " << h_z[k] << "\n";
      throw std::runtime_error("RESULT MISMATCH");
    }
  }
  std::cout << ", " << min_ms << " ms"
            << ", " << 3*n*sizeof(T)/min_ms*1e-6 << " GB/s"
            << "\n";

  delete[] h_x;
  delete[] h_y;
  delete[] h_z;
  CHECK_CUDA(hipFree(x));
  CHECK_CUDA(hipFree(y));
  CHECK_CUDA(hipEventDestroy(cstart));
  CHECK_CUDA(hipEventDestroy(cend));
  CHECK_CUDA(hipStreamDestroy(cstream));

}

int main(int argc, const char** argv)
{

  static constexpr int REPETITIONS = 3;
  using DATA_TYPE = int;

  const int dev=0;
  unsigned int n1 = 0;
  unsigned int n2 = 0;
  if(argc>=2)
    n1 = atoi(argv[1]);
  if(n1<2)
    n1 = 1<<28;
  if(argc==3) // range
    n2 = atoi(argv[2]);
  if(n2<n1)
    n2 = n1;

  print_header("saxpy-mono",n1,n2);

  for(unsigned n=n1; n<=n2; n<<=1) {
    saxpy<DATA_TYPE, REPETITIONS, 64>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 128>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 256>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 512>(n, dev);
    saxpy<DATA_TYPE, REPETITIONS, 1024>(n, dev);
  }

  CHECK_CUDA( hipDeviceReset() );
  return 0;
}
